#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define THREAD_COUNT 1024

using namespace std;

void readGraph(int &vertexCount, int &edgeCount, int &maxDegree, int **adjList, int **adjListPointers)
{
    int u, v;
    cin >> vertexCount >> edgeCount;

    vector<int> *adj = new vector<int>[vertexCount];
    for (int i = 0; i < edgeCount; i++)
    {
        cin >> u >> v;
        adj[u].push_back(v);
        adj[v].push_back(u);
    }

    *adjList = new int[edgeCount * 2 + 1];
    *adjListPointers = new int[vertexCount + 1];
    int pos = 0;
    for (int i = 0; i < vertexCount; i++)
    {
        (*adjListPointers)[i] = pos;
        for (int j = 0; j < adj[i].size(); j++)
        {
            (*adjList)[pos++] = adj[i][j];
        }
    }
    (*adjListPointers)[vertexCount] = pos;

    maxDegree = INT_MIN;
    for (int i = 0; i < vertexCount; i++)
    {
        maxDegree = max(maxDegree, (int)adj[i].size());
    }

    delete[] adj;

    return;
}

__global__
void assignColors(int vertexCount, int *adjList, int *adjListPointers, int *colors, bool *conflicts, int maxDegree)
{
    int vertex = blockIdx.x * blockDim.x + threadIdx.x;
    if (vertex >= vertexCount || !conflicts[vertex])
    {
        return;
    }

    int maxColors = maxDegree + 1;

    int *forbidden = new int[maxColors/32 + 1];
    if (forbidden == NULL)
    {
        printf("Memory allocation failed\n");
        return;
    }
    memset(forbidden, 0, sizeof(int) * (maxColors/32 + 1));

    for (int i = adjListPointers[vertex]; i < adjListPointers[vertex + 1]; i++)
    {
        int neighbor = adjList[i];
        int ind = colors[neighbor] % 32;
        forbidden[colors[neighbor] / 32] |= (1 << ind);
    }

    for (int c = 1; c < maxColors; c++)
    {
        int ind = c % 32;
        if ((forbidden[c / 32] & (1 << ind)) == 0)
        {
            colors[vertex] = c;
            break;
        }
    }

    delete[] forbidden;
}

void assignColorsWrapper(int vertexCount, int *adjList, int *adjListPointers, int *colors, bool *conflicts, int maxDegree)
{
    assignColors<<<(vertexCount + THREAD_COUNT - 1) / THREAD_COUNT, THREAD_COUNT>>>(vertexCount, adjList, adjListPointers, colors, conflicts, maxDegree);
    hipDeviceSynchronize();
}

__global__
void detectConflicts(int vertexCount, int *adjList, int *adjListPointers, int *colors, bool *conflicts, bool *conflictExists)
{
    int vertex = blockIdx.x * blockDim.x + threadIdx.x;
    if (vertex >= vertexCount)
    {
        return;
    }

    conflicts[vertex] = false;

    for (int i = adjListPointers[vertex]; i < adjListPointers[vertex + 1]; i++)
    {
        int neighbor = adjList[i];
        if (colors[vertex] == colors[neighbor] && neighbor < vertex)
        {
            conflicts[vertex] = true;
            *conflictExists = true;
        }
    }
}

bool detectConflictsWrapper(int vertexCount, int *adjList, int *adjListPointers, int *colors, bool *conflicts)
{
    bool *device_conflictExists;
    hipMalloc(&device_conflictExists, sizeof(bool));
    hipMemset(device_conflictExists, 0, sizeof(bool));

    detectConflicts<<<(vertexCount + THREAD_COUNT - 1) / THREAD_COUNT, THREAD_COUNT>>>(vertexCount, adjList, adjListPointers, colors, conflicts, device_conflictExists);
    hipDeviceSynchronize();

    bool host_conflictExists = false;
    hipMemcpy(&host_conflictExists, device_conflictExists, sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(device_conflictExists);

    return host_conflictExists;
}

int *graphColoring(int vertexCount, int *adjList, int *adjListPointers, int maxDegree)
{
    bool *host_conflicts = new bool[vertexCount];
    bool *device_conflicts;
    int *host_colors = new int[vertexCount];
    int *device_colors;

    memset(host_colors, 0, sizeof(int) * vertexCount);
    memset(host_conflicts, true, sizeof(bool) * vertexCount);

    hipMalloc((void**)&device_colors, sizeof(int) * vertexCount);
    hipMemcpy(device_colors, host_colors, sizeof(int) * vertexCount, hipMemcpyHostToDevice);
    hipMalloc((void**)&device_conflicts, sizeof(bool) * vertexCount);
    hipMemcpy(device_conflicts, host_conflicts, sizeof(bool) * vertexCount, hipMemcpyHostToDevice);

    do {
        assignColorsWrapper(vertexCount, adjList, adjListPointers, device_colors, device_conflicts, maxDegree);
    } while (detectConflictsWrapper(vertexCount, adjList, adjListPointers, device_colors, device_conflicts));

    hipMemcpy(host_colors, device_colors, sizeof(int) * vertexCount, hipMemcpyDeviceToHost);

    delete[] host_conflicts;
    hipFree(device_colors);
    hipFree(device_conflicts);

    return host_colors;
}

int main(int argc, char *argv[])
{
    using chrono::high_resolution_clock;
	using chrono::duration;

    int vertexCount, edgeCount, maxDegree;
    int *adjList = NULL, *adjListPointers = NULL;
    int *device_adjList, *device_adjListPointers;
    string in_file_name = argv[1];
    string out_file_name = argv[2];

    auto readGraphStart = high_resolution_clock::now();

    freopen(in_file_name.c_str(), "r", stdin);
    readGraph(vertexCount, edgeCount, maxDegree, &adjList, &adjListPointers);

    auto parallelStart = high_resolution_clock::now();

    hipMalloc(&device_adjList, sizeof(int) * (edgeCount * 2 + 1));
    hipMemcpy(device_adjList, adjList, sizeof(int) * (edgeCount * 2 + 1), hipMemcpyHostToDevice);
    hipMalloc(&device_adjListPointers, sizeof(int) * (vertexCount + 1));
    hipMemcpy(device_adjListPointers, adjListPointers, sizeof(int) * (vertexCount + 1), hipMemcpyHostToDevice);

    int *coloring = graphColoring(vertexCount, device_adjList, device_adjListPointers, maxDegree);

    auto parallelEnd = high_resolution_clock::now();

    cout << endl;

    int numColorsUsed = 0;
    for (int i = 0; i < vertexCount; i++)
    {
        numColorsUsed = max(numColorsUsed, coloring[i]);
    }
    cout << "Number of colors used: " << numColorsUsed << endl;

    int numConflicts = 0;
    for (int u = 0; u < vertexCount; u++)
    {
        for (int i = adjListPointers[u]; i < adjListPointers[u + 1]; i++)
        {
            int v = adjList[i];
            if (coloring[u] == coloring[v] && u <= v)
            {
                cout << "Conflict found between " << u << " and " << v << endl;
                numConflicts++;
            }
        }
    }
    cout << "Number of conflicts: " << numConflicts << endl;
    cout << endl;

    duration<double, milli> parallelTime = parallelEnd - parallelStart;
    duration<double, milli> totalTime = parallelEnd - readGraphStart;

    cout << "Parallel time: " << parallelTime.count() << " ms" << endl;
    cout << "Total time: " << totalTime.count() << " ms" << endl;

    freopen(out_file_name.c_str(), "w", stdout);
    for (int i = 0; i < vertexCount; i++)
    {
        cout << coloring[i] << endl;
    }

    delete[] coloring, adjList, adjListPointers;
    hipFree(device_adjList);
    hipFree(device_adjListPointers);

    return 0;
}
